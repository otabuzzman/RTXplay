#include "hip/hip_runtime.h"
//
// Copyright (c) 2019, NVIDIA CORPORATION. All rights reserved.
//
// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions
// are met:
//  * Redistributions of source code must retain the above copyright
//    notice, this list of conditions and the following disclaimer.
//  * Redistributions in binary form must reproduce the above copyright
//    notice, this list of conditions and the following disclaimer in the
//    documentation and/or other materials provided with the distribution.
//  * Neither the name of NVIDIA CORPORATION nor the names of its
//    contributors may be used to endorse or promote products derived
//    from this software without specific prior written permission.
//
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
// EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
// IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
// PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
// CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
// EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
// PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
// PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
// OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
// (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
// OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
//

#include <optix.h>

#include "camera.h"
#include "optics.h"

#include "optixTriangle.h"

using V::operator+ ;
using V::operator- ;
using V::operator* ;

extern "C" { __constant__ LpGeneral lp_general ; }

static __forceinline__ __device__ uchar4 sRGB( const float3& color ) {
	return make_uchar4(
		(unsigned char) ( util::clamp( color.x, .0f, 1.f )*255.f+.5f ),
		(unsigned char) ( util::clamp( color.y, .0f, 1.f )*255.f+.5f ),
		(unsigned char) ( util::clamp( color.z, .0f, 1.f )*255.f+.5f ), 255u ) ;
}

extern "C" __global__ void __raygen__camera() {
	const uint3 idx = optixGetLaunchIndex() ;
	const uint3 dim = optixGetLaunchDimensions() ;

	// set pixel index
	const unsigned int pix = dim.x*idx.y+idx.x ;

	// initialize random number generator
	hiprandState state ;
	hiprand_init( 4711, pix, 0, &state ) ;

	// prepare RNG state pointer for payload transmission
	unsigned int sh = reinterpret_cast<unsigned long long>( &state )>>32 ;
	unsigned int sl = reinterpret_cast<unsigned long long>( &state )&0x00000000ffffffff ;

	// transform x/y pixel coords (range 0/0 to w/h)
	// into s/t viewport coords (range -1/-1 to 1/1)
	const float s = 2.f*static_cast<float>( idx.x )/static_cast<float>( dim.x )-1.f ;
	const float t = 2.f*static_cast<float>( idx.y )/static_cast<float>( dim.y )-1.f ;

	// get Camera class instance from SBT
	const Camera* camera  = reinterpret_cast<Camera*>( optixGetSbtDataPointer() ) ;

	float3 ori, dir ;
	camera->ray( s, t, ori, dir, &state ) ;

	// initialize depth count
	unsigned int depth = 1 ;

	// trace into scene
	unsigned int r, g, b ;
	optixTrace(
			lp_general.as_handle,
			ori,
			dir,
			1e-3f,                      // Min intersection distance
			1e16f,                      // Max intersection distance
			0.f,                        // rayTime -- used for motion blur
			OptixVisibilityMask( 255 ), // Specify always visible
			OPTIX_RAY_FLAG_NONE,
			0,                          // SBT offset   -- See SBT discussion
			1,                          // SBT stride   -- See SBT discussion
			0,                          // missSBTIndex -- See SBT discussion
			r, g, b,
			sh, sl,
			depth
			) ;

	const float3 color = make_float3(
			__uint_as_float( r ),
			__uint_as_float( g ),
			__uint_as_float( b )
			) ;

	lp_general.image[pix] = sRGB( color ) ;
}

extern "C" __global__ void __miss__ambient() {
	// get ambient color from MS program group's SBT record
	const float3 ambient = *reinterpret_cast<float3*>( optixGetSbtDataPointer() ) ;

	// get this ray's direction from OptiX and normalize
	const float3 unit    = V::unitV( optixGetWorldRayDirection() ) ;

	const float t        = .5f*( unit.y+1.f ) ;
	const float3 white   = { 1.f, 1.f, 1.f } ;
	const float3 color   = ( 1.f-t )*white+t*ambient ;

	optixSetPayload_0( __float_as_uint( color.x ) ) ;
	optixSetPayload_1( __float_as_uint( color.y ) ) ;
	optixSetPayload_2( __float_as_uint( color.z ) ) ;
}

extern "C" __global__ void __closesthit__diffuse() {
	unsigned int depth = optixGetPayload_5() ;

	if ( lp_general.depth>depth ) {
		const Optics optics = *reinterpret_cast<Optics*>( optixGetSbtDataPointer() ) ;

		const int   prix = optixGetPrimitiveIndex() ;
		const uint3 trix = optics.ices[prix] ;

		const float3 A = optics.vces[trix.x] ;
		const float3 B = optics.vces[trix.y] ;
		const float3 C = optics.vces[trix.z] ;

		const float2 bary = optixGetTriangleBarycentrics() ;
		const float u = bary.x ;
		const float v = bary.y ;
		const float w = 1.f-u-v ;

		// calculate primitive hit point
		const float3 ori = w*A+u*B+v*C ;

		// calculate primitive normal
		const float3 d = optixGetWorldRayDirection() ;
		float3 N = V::unitV( V::cross( B-A, C-A ) ) ;
		if ( V::dot( d, N )>0.f )
			N = -N ;

		// retrieve and assemble hiprandState pointer from payload
		unsigned int sh = optixGetPayload_3() ;
		unsigned int sl = optixGetPayload_4() ;
		hiprandState* state = reinterpret_cast<hiprandState*>( static_cast<unsigned long long>( sh )<<32|sl ) ;

		const float3 dir = N+V::rndVon1sphere( state ) ;

		unsigned int r, g, b ;
		optixTrace(
				lp_general.as_handle,
				ori,
				dir,
				1e-3f,                      // Min intersection distance
				1e16f,                      // Max intersection distance
				0.f,                        // rayTime -- used for motion blur
				OptixVisibilityMask( 255 ), // Specify always visible
				OPTIX_RAY_FLAG_NONE,
				0,                          // SBT offset   -- See SBT discussion
				1,                          // SBT stride   -- See SBT discussion
				0,                          // missSBTIndex -- See SBT discussion
				r, g, b,
				sh, sl,
				++depth
				) ;

		const float3 albedo = optics.diffuse.albedo ;
		optixSetPayload_0( __float_as_uint( albedo.x*__uint_as_float( r ) ) ) ;
		optixSetPayload_1( __float_as_uint( albedo.y*__uint_as_float( g ) ) ) ;
		optixSetPayload_2( __float_as_uint( albedo.z*__uint_as_float( b ) ) ) ;
	} else {
		optixSetPayload_0( 0u ) ;
		optixSetPayload_1( 0u ) ;
		optixSetPayload_2( 0u ) ;
	}
}

extern "C" __global__ void __closesthit__reflect() {
    // When built-in triangle intersection is used, a number of fundamental
    // attributes are provided by the OptiX API, indlucing barycentric coordinates.
    const float2 barycentrics = optixGetTriangleBarycentrics();
    const float3 color = { barycentrics.x, barycentrics.y, .5f };

	optixSetPayload_0( __float_as_uint( color.x ) ) ;
	optixSetPayload_1( __float_as_uint( color.y ) ) ;
	optixSetPayload_2( __float_as_uint( color.z ) ) ;
}

extern "C" __global__ void __closesthit__refract() {
    // When built-in triangle intersection is used, a number of fundamental
    // attributes are provided by the OptiX API, indlucing barycentric coordinates.
    const float2 barycentrics = optixGetTriangleBarycentrics();
    const float3 color = { barycentrics.x, barycentrics.y, 0.f };

	optixSetPayload_0( __float_as_uint( color.x ) ) ;
	optixSetPayload_1( __float_as_uint( color.y ) ) ;
	optixSetPayload_2( __float_as_uint( color.z ) ) ;
}
